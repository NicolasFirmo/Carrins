#include <hip/hip_runtime.h>
 // threadIdx
#include <cstdio>

#pragma diag_suppress integer_sign_change

static unsigned short* d_in;
static unsigned char* d_out;

static unsigned int h_Width;
static unsigned int h_Height;

static unsigned int h_BlockWidth;
static unsigned int h_BlockHeight;

#define THREAD_TOTAL_X_LEN 12
#define THREAD_AUX_X_LEN 4
#define THREAD_WORKING_X_LEN (THREAD_TOTAL_X_LEN - THREAD_AUX_X_LEN)

#define THREAD_TOTAL_Y_LEN 12
#define THREAD_AUX_Y_LEN 4
#define THREAD_WORKING_Y_LEN (THREAD_TOTAL_Y_LEN - THREAD_AUX_Y_LEN)

#define OFFSET(x,y) sIdx + y * THREAD_TOTAL_X_LEN + x

__global__ void Sobel(const unsigned short* in,unsigned char* out, const unsigned int width, const unsigned int height)
{
	extern __shared__ float s[];

	const unsigned int xPos = (blockIdx.x * THREAD_WORKING_X_LEN + threadIdx.x) - (THREAD_AUX_X_LEN / 2);
	const unsigned int yPos = (blockIdx.y * THREAD_WORKING_Y_LEN + threadIdx.y) - (THREAD_AUX_Y_LEN / 2);
	const unsigned int inPos = (xPos + yPos * width);
	const unsigned int sIdx = (threadIdx.x + threadIdx.y * THREAD_TOTAL_X_LEN);
	unsigned int outIt = inPos * 4;

	if (xPos < width && yPos < height)
		s[sIdx] = in[inPos] / float(USHRT_MAX);
	else
		s[sIdx] = 0.0f;

	__syncthreads();

	if ((threadIdx.x - (THREAD_AUX_X_LEN / 2)) < THREAD_WORKING_X_LEN && (threadIdx.y - (THREAD_AUX_X_LEN / 2)) < THREAD_WORKING_Y_LEN)
	{
		const float sobelX = (
													-1 * s[OFFSET(-2,-2)] -2 * s[OFFSET(-1,-2)] +0 * s[OFFSET(0,-2)] +2 * s[OFFSET(1,-2)] +1 * s[OFFSET(2,-2)]
													-1 * s[OFFSET(-2,-1)] -2 * s[OFFSET(-1,-1)] +0 * s[OFFSET(0,-1)] +2 * s[OFFSET(1,-1)] +1 * s[OFFSET(2,-1)]
													-2 * s[OFFSET(-2, 0)] -4 * s[OFFSET(-1, 0)] +0 * s[OFFSET(0, 0)] +4 * s[OFFSET(1, 0)] +2 * s[OFFSET(2, 0)]
													-1 * s[OFFSET(-2, 1)] -2 * s[OFFSET(-1, 1)] +0 * s[OFFSET(0, 1)] +2 * s[OFFSET(1, 1)] +1 * s[OFFSET(2, 1)]
													-1 * s[OFFSET(-2, 2)] -2 * s[OFFSET(-1, 2)] +0 * s[OFFSET(0, 2)] +2 * s[OFFSET(1, 2)] +1 * s[OFFSET(2, 2)]
												)*(512/8)/18;

		const float sobelY =	(
													+1 * s[OFFSET(-2,-2)] +1 * s[OFFSET(-1,-2)] +2 * s[OFFSET(0,-2)] +1 * s[OFFSET(1,-2)] +1 * s[OFFSET(2,-2)]
													+2 * s[OFFSET(-2,-1)] +2 * s[OFFSET(-1,-1)] +4 * s[OFFSET(0,-1)] +2 * s[OFFSET(1,-1)] +2 * s[OFFSET(2,-1)]
													+0 * s[OFFSET(-2, 0)] +0 * s[OFFSET(-1, 0)] +0 * s[OFFSET(0, 0)] +0 * s[OFFSET(1, 0)] +0 * s[OFFSET(2, 0)]
													-2 * s[OFFSET(-2, 1)] -2 * s[OFFSET(-1, 1)] -4 * s[OFFSET(0, 1)] -2 * s[OFFSET(1, 1)] -2 * s[OFFSET(2, 1)]
													-1 * s[OFFSET(-2, 2)] -1 * s[OFFSET(-1, 2)] -2 * s[OFFSET(0, 2)] -1 * s[OFFSET(1, 2)] -1 * s[OFFSET(2, 2)]
												)*(512/8)/18;

		const float gradientLen = sqrt(sobelX*sobelX + sobelY*sobelY + 1.0f);

		const unsigned char xLen = -(sobelX * 128)/gradientLen + 128;
		const unsigned char yLen = -(sobelY * 128)/gradientLen + 128;
		const unsigned char zLen = (UCHAR_MAX)/gradientLen;

		out[outIt++] = xLen;
		out[outIt++] = yLen;
		out[outIt++] = zLen;
		out[outIt] = 255;
	}
}

void InitBumpToNormalMap(const unsigned int width, const unsigned int height)
{
	h_Width = width;
	h_Height = height;

	h_BlockWidth = (h_Width / THREAD_WORKING_X_LEN);
	h_BlockHeight = (h_Height / THREAD_WORKING_Y_LEN);

	hipMalloc(&d_in, h_Width * h_Height * 1 * sizeof(unsigned short));
	hipMalloc(&d_out, h_Width * h_Height * 4 * sizeof(unsigned char));
}

int KernelBumpToNormalMap(const unsigned short* h_in_img,unsigned char* h_out_img)
{
	hipError_t error = hipMemcpy(d_in, h_in_img, h_Width * h_Height * 1 * sizeof(unsigned short), hipMemcpyHostToDevice);
	if (error != hipSuccess)
		return error;

	Sobel<<<dim3(h_BlockWidth, h_BlockHeight, 1), dim3(THREAD_TOTAL_X_LEN, THREAD_TOTAL_Y_LEN, 1), THREAD_TOTAL_X_LEN * THREAD_TOTAL_Y_LEN * sizeof(float)>>>(d_in, d_out, h_Width, h_Height);
	error = hipGetLastError();
	if (error != hipSuccess)
		return error;

	error = hipMemcpy(h_out_img, d_out, h_Width * h_Height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
		return error;

	return hipSuccess;
}

void ShutdownBumpToNormalMap()
{
	hipFree(d_in);
	hipFree(d_out);
}

#pragma diag_default integer_sign_change